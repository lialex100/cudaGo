
#include <hip/hip_runtime.h>
const size_t NTB = 256;
const size_t EXT = 8;
#define divCeil(a, b) (((a) + (b) - 1) / (b))
struct Ctx {
    float *x, *y, *r;
    size_t n;
};

__global__ void devDot(float *x, float *y, size_t n, float *r) {
    __shared__ float rb[NTB];
    size_t itb = threadIdx.x;
    size_t i = blockIdx.x * blockDim.x * EXT + itb;
    float s = 0.0;
    for (size_t j = 0; j < EXT && i < n; j++, i += blockDim.x) {
        s += x[i] * y[i];
    }

    rb[itb] = s;
    __syncthreads();
    for (size_t i = NTB >> 1; i != 0; i >>= 1) {
        if (itb < i) rb[itb] += rb[itb + i];
        __syncthreads();
    }
    if (0 == itb) r[blockIdx.x] = rb[0];
}

extern "C" __declspec(dllexport) void getInputs(Ctx *ctx, float **px, float **py) {
    *px = ctx->x;
    *py = ctx->y;
}

extern "C" __declspec(dllexport) void init(Ctx **p, size_t n) {
    Ctx *ctx = (Ctx *)malloc(sizeof(Ctx));
    ctx->n = n;
    size_t sz = sizeof(float) * n;
    hipMallocManaged(&(ctx->x), sz);
    hipMallocManaged(&(ctx->y), sz);
    hipMallocManaged(&(ctx->r), sizeof(float) * divCeil(n, NTB) / EXT);
    *p = ctx;
}

extern "C" __declspec(dllexport) void deinit(Ctx *ctx) {
    hipFree(ctx->x);
    hipFree(ctx->y);
    hipFree(ctx->r);
    free(ctx);
}
extern "C" __declspec(dllexport) void dot(Ctx *ctx, float *r) {
    size_t nb = divCeil(ctx->n, NTB) / EXT;
    float *rd = ctx->r;
    devDot<<<nb, NTB>>>(ctx->x, ctx->y, ctx->n, rd);
    hipDeviceSynchronize();
    float s = 0.0;
    for (size_t i = 0; i < nb; i++) s += rd[i];
    *r = s;
}